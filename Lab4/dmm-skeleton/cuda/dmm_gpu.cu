#include "hip/hip_runtime.h"
/*
 *  dmm_gpu.cu -- Template for DMM GPU kernels
 *
 *  Copyright (C) 2020, Computing Systems Laboratory (CSLab)
 *  Copyright (C) 2020, Athena Elafrou/Petros Anastasiadis
 */

#include "dmm.h"
#include "stdio.h"
#include <hipblas.h>

/*
 *  Naive kernel
 */
__global__ void dmm_gpu_naive(const value_t *A, const value_t *B, value_t *C,
                              const size_t M, const size_t N, const size_t K) {

    int tidx,tidy;
    value_t sum = 0;
    tidx = blockDim.x * blockIdx.x + threadIdx.x;
    tidy = blockDim.y * blockIdx.y + threadIdx.y;

    if (tidx<N && tidy<M)
    {
        for (int i=0; i<K; i++)
        {
            sum = sum + A[tidy*K+i]*B[tidx+N*i];
        }

        C[tidy*N+tidx] = sum;
    } 
}

/*
 *  Coalesced memory acceses of A.
 */
__global__ void dmm_gpu_coalesced_A(const value_t *A, const value_t *B,
				    value_t *C, const size_t M, const size_t N,
				    const size_t K) {
  /*
   * FILLME: fill the code.
   */
    __shared__ value_t A_shared[TILE_Y][TILE_X];
    
    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int col = bx * TILE_X + tx;
    int row = by * TILE_Y + ty;

    value_t sum = 0;

    for (int i=0; i<(K+TILE_X-1)/TILE_X; i++) {
        A_shared[ty][tx] = A[row*K + (i*TILE_X+tx)];

        __syncthreads();

        for(int k=0; k<TILE_X; k++) {
            sum += A_shared[ty][k]*B[(i*TILE_X+k)*N+col];
        }
    }

    C[row*N+col] = sum;
}

/*
 *  Reduced memory accesses.
 */
__global__ void dmm_gpu_reduced_global(const value_t *A, const value_t *B, value_t *C,
				       const size_t M, const size_t N, const size_t K) {
  /*
   * FILLME: fill the code.
   */
    __shared__ value_t A_shared[TILE_Y][TILE_X];
    __shared__ value_t B_shared[TILE_Y][TILE_X];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;
    
    int col = bx * TILE_X + tx;
    int row = by * TILE_Y + ty;

    value_t sum = 0;

    for (int i=0; i<(K+TILE_X-1)/TILE_X; i++) {
        A_shared[ty][tx] = A[row*K + i*TILE_X+tx];
        B_shared[ty][tx] = B[col + (i*TILE_Y+ty)*N];

        __syncthreads();

        for(int k=0; k<TILE_X; k++) {
            sum += A_shared[ty][k]*B_shared[k][tx];
        }
        __syncthreads();
    }

    C[row*N+col] = sum;
}

/*
 *  Use of cuBLAS
 */
void dmm_gpu_cublas(const value_t *A, const value_t *B, value_t *C,
		    const size_t M, const size_t N, const size_t K) {
  /*
   * source: https://solarianprogrammer.com/2012/05/31/matrix-multiplication-cuda-cublas-hiprand-thrust/
   */

    int lda = N;
    int ldb = K;
    int ldc = N;

    const float alf = 1;
    const float bet = 0;
    const float *alpha = &alf;
    const float *beta = &bet;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, alpha, A, lda, B, ldb, beta, C, ldc);

    hipblasDestroy(handle);
}
